#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include "Header.h"

#define NUM_THREADS_IN_BLOCK 1000

hipError_t refreshPointsInitCuda(Point *points, const int n, const double t);
hipError_t Error(Point* dev_points);

__global__ void refreshPointsKernel(Point *points, const double t)
{
	int i = blockIdx.x;
    int j = threadIdx.x;
	points[NUM_THREADS_IN_BLOCK*i + j].x = points[NUM_THREADS_IN_BLOCK*i + j].x + t*points[NUM_THREADS_IN_BLOCK*i + j].vx;
	points[NUM_THREADS_IN_BLOCK*i + j].y = points[NUM_THREADS_IN_BLOCK*i + j].y + t*points[NUM_THREADS_IN_BLOCK*i + j].vy;
}

int cudaRefreshPoints(Point* points, int n, double dt)
{

    // Add vectors in parallel.
    hipError_t cudaStatus = refreshPointsInitCuda(points, n, dt);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t refreshPointsInitCuda(Point *points, const int n, const double t)
{
    Point *dev_points = 0;
    hipError_t cudaStatus;

	int numOfBlocks;
	if (n % NUM_THREADS_IN_BLOCK == 0)
		numOfBlocks = n / NUM_THREADS_IN_BLOCK;
	else
		numOfBlocks = n / NUM_THREADS_IN_BLOCK + 1;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		Error(dev_points);
		return cudaStatus;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_points, n * sizeof(Point));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
		Error(dev_points);
		return cudaStatus;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_points, points, n * sizeof(Point), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
		Error(dev_points);
		return cudaStatus;
    }

    // Launch a kernel on the GPU with one thread for each element.
	refreshPointsKernel <<<numOfBlocks, NUM_THREADS_IN_BLOCK>>>(dev_points, t);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		Error(dev_points);
		return cudaStatus;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		Error(dev_points);
		return cudaStatus;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(points, dev_points, n * sizeof(Point), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
		Error(dev_points);
		return cudaStatus;
    }
	return Error(dev_points);
}

hipError_t Error(Point* dev_points)
{
	hipError_t cudaStatus;
	cudaStatus = hipFree(dev_points);
	if (cudaStatus != hipSuccess)
		fprintf(stderr, "hipFree failed!");
	
	return cudaStatus;
}