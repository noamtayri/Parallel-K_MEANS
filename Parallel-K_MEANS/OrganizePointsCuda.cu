#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include "Header.h"

#define NUM_THREADS_IN_BLOCK 1000

hipError_t pointsOrgenaizeCuda(Cluster* clusters, Point *points, const int n, const int k, bool *flag);
hipError_t Error(Point* dev_points, Cluster* dev_clusters, bool* dev_flags);

__device__ double distanceCuda(double x1, double y1, double x2, double y2)
{
	return sqrt(pow(x1 - x2, 2) + pow(y1 - y2, 2));
}

__global__ void organizePointsKernel(Cluster *clusters, Point *points, bool *flags, const int k, const int n)
{
	double min = DBL_MAX;
	int minIdx;
	int i = blockIdx.x;
	int j = threadIdx.x;
	int idx = NUM_THREADS_IN_BLOCK * i + j;
	if (idx < n)
	{
		for (int l = 0; l < k; l++)
		{
			double tempDistance = distanceCuda(points[idx].x, points[idx].y, clusters[l].centerX, clusters[l].centerY);
			if (tempDistance < min) {
				minIdx = l;
				min = tempDistance;
			}
		}
		if (points[idx].myCluster != minIdx)
			flags[idx] = true;
		points[idx].myCluster = minIdx;
	}
}

int cudaOrganizePoints(Cluster* clusters, Point* points, int n, int k, bool *flag)
{
	*flag = false;

	// Add vectors in parallel.
	hipError_t cudaStatus = pointsOrgenaizeCuda(clusters,points, n, k, flag);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t pointsOrgenaizeCuda(Cluster* clusters, Point *points, const int n, const int k, bool *flag)
{
	Cluster *dev_clusters = 0;
	Point *dev_points = 0;
	hipError_t cudaStatus;
	int numOfBlocks;
	if(n % NUM_THREADS_IN_BLOCK == 0)
		numOfBlocks = n / NUM_THREADS_IN_BLOCK;
	else
		numOfBlocks = n / NUM_THREADS_IN_BLOCK + 1;
	bool* dev_flags;
	bool* flags = (bool*)malloc(n * sizeof(bool));
	for (int i = 0; i < n; i++)
	{
		flags[i] = false;
	}
	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		//goto Error;
		Error(dev_points, dev_clusters, dev_flags);
		return cudaStatus;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_points, n * sizeof(Point));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		Error(dev_points, dev_clusters, dev_flags);
		return cudaStatus;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_clusters, k * sizeof(Cluster));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		Error(dev_points, dev_clusters, dev_flags);
		return cudaStatus;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_flags, n * sizeof(bool));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		Error(dev_points, dev_clusters, dev_flags);
		return cudaStatus;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_flags, flags, n * sizeof(bool), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		Error(dev_points, dev_clusters, dev_flags);
		return cudaStatus;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_points, points, n * sizeof(Point), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		Error(dev_points, dev_clusters, dev_flags);
		return cudaStatus;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_clusters, clusters, k * sizeof(Cluster), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		Error(dev_points, dev_clusters, dev_flags);
		return cudaStatus;
	}

	// Launch a kernel on the GPU with one thread for each element.
	organizePointsKernel <<<numOfBlocks, NUM_THREADS_IN_BLOCK >>>(dev_clusters, dev_points, dev_flags, k,n);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		Error(dev_points, dev_clusters, dev_flags);
		return cudaStatus;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		Error(dev_points, dev_clusters, dev_flags);
		return cudaStatus;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(points, dev_points, n * sizeof(Point), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		Error(dev_points, dev_clusters, dev_flags);
		return cudaStatus;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(clusters, dev_clusters, k * sizeof(Cluster), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		Error(dev_points, dev_clusters, dev_flags);
		return cudaStatus;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(flags, dev_flags, n * sizeof(bool), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		Error(dev_points, dev_clusters, dev_flags);
		return cudaStatus;
	}

	for (int i = 0; i < n; i++)
	{
		if (flags[i] == true)
		{
			*flag = true;
			break;
		}
	}
	return Error(dev_points, dev_clusters, dev_flags);
}

hipError_t Error(Point* dev_points, Cluster* dev_clusters, bool* dev_flags)
{
	hipError_t cudaStatus;
	cudaStatus = hipFree(dev_points);
	if (cudaStatus != hipSuccess) 
		fprintf(stderr, "hipFree failed!");
	cudaStatus = hipFree(dev_clusters);
	if (cudaStatus != hipSuccess) 
		fprintf(stderr, "hipFree failed!");
	cudaStatus = hipFree(dev_flags);
	if (cudaStatus != hipSuccess) 
		fprintf(stderr, "hipFree failed!");

	return cudaStatus;
}
